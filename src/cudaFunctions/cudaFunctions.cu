#include "cudaFunctions.cuh"
#include "hip/hip_runtime.h"
#include "env.cuh"

// function  from hipMemcpyKind to str
const char *cudaMemcpyKindToStr(hipMemcpyKind kind)
{
	switch (kind)
	{
	case hipMemcpyHostToHost:
		return "hipMemcpyHostToHost";
	case hipMemcpyHostToDevice:
		return "hipMemcpyHostToDevice";
	case hipMemcpyDeviceToHost:
		return "hipMemcpyDeviceToHost";
	case hipMemcpyDeviceToDevice:
		return "hipMemcpyDeviceToDevice";
	default:
		return "hipMemcpyDefault";
	}
}


void *cudaMallocX(size_t size)
{
	void *ret;
	hipError_t err = hipMalloc(&ret,size);
	if (err != hipSuccess)
	{
		spdlog::error("Error allocating memory in GPU: {}", hipGetErrorString(err));
		std::exit(1);
	}
	return ret;
}

void *cudaMallocPitchX(size_t *pitch, size_t width, size_t height)
{
	void *ret;
	hipError_t err = hipMallocPitch(&ret, pitch, width, height);
	if (err != hipSuccess)
	{
		spdlog::error("Error allocating memory in GPU: {}", hipGetErrorString(err));
		std::exit(1);
	}
	return ret;
}

void cudaMemcpy2DX(void *dst, size_t dpitch, const void *src, size_t spitch,
				   size_t width, size_t height, hipMemcpyKind kind)
{
	hipError_t err = hipMemcpy2D(dst, dpitch, src, spitch, width, height, kind);
	if (err != hipSuccess)
	{
		spdlog::error("Error copying memory in {}: {}", cudaMemcpyKindToStr(kind), hipGetErrorString(err));
		std::exit(1);
	}
}

void cudaFreeX(void *ptr)
{
	hipError_t err = hipFree(ptr);
	if (err != hipSuccess)
	{
		spdlog::error("Error freeing memory in GPU: {}", hipGetErrorString(err));
		std::exit(1);
	}
}

void cudaDeviceSynchronizeX()
{
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess)
	{
		spdlog::error("Error synchronizing GPU: {}", hipGetErrorString(err));
		std::exit(1);
	}
}

void cudaMemcpyX(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
	hipError_t err = hipMemcpy(dst, src, count, kind);
	if (err != hipSuccess)
	{
		spdlog::error("Error copying memory in {}: {}", cudaMemcpyKindToStr(kind), hipGetErrorString(err));
		std::exit(1);
	}
}
